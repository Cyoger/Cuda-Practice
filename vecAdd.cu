#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

/*
Kernel function to add vectors.
__global__ is used to mark function as a kernel (callable from CPU, executed on GPU)
 */
__global__ void vecAddKernel(float *A, float *B, float *C, int n){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n){
		C[i] = A[i] + B[i];
	}
}

void vecAdd(float *A_h, float *B_h, float *C_h, int n){
	int size = n * sizeof(float);
	float *A_d, *B_d, *C_d;
	hipMalloc((void**) &A_d, size);
    hipMalloc((void**) &B_d, size);
	hipMalloc((void**) &C_d, size);
	
	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = ceil((float)n / threadsPerBlock); // calculates number of blocks needed to have a thread for every element in the array 
	vecAddKernel<<<blocksPerGrid, threadsPerBlock>>>(A_d, B_d, C_d, n);

	hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	
}

int main() {
	float A[4] = {1, 2, 3, 4};
	float B[4] = {1, 2, 3, 4};
	float C[4];
	vecAdd(A, B, C, 4);
	for (int i = 0; i < 4; i++) {
		printf("%f ", C[i]);
	}
	printf("\n");
	return 0;
}